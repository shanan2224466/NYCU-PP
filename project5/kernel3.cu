#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float stepX, float stepY, float lowerX, float lowerY, int thre_width, int thre_height, int* d_img, int maxIterations, int pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    // Only change the coordinate thisX, thisY of each thread with thre_width and thre_height.
    for (int x = 0; x < thre_width; x++)
    {
        for (int y = 0; y < thre_height; y++)
        {
            int thisX = ((blockIdx.x * blockDim.x + threadIdx.x) + x * 800);
            int thisY = ((blockIdx.y * blockDim.y + threadIdx.y) + y * 600);
            float c_re = lowerX + thisX * stepX;
            float c_im = lowerY + thisY * stepY;

            float z_re = c_re, z_im = c_im;
            int i;
            for (i = 0; i < maxIterations; ++i)
            {
                if (z_re * z_re + z_im * z_im > 4.f)
                    break;

                float new_re = z_re * z_re - z_im * z_im;
                float new_im = 2.f * z_re * z_im;
                z_re = c_re + new_re;
                z_im = c_im + new_im;
            }
            d_img[thisX + thisY * pitch / 4] = i;
        }
    }
    return;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *d_img;
    int *h_img;
    size_t pitch;

    // Allocate pinned memory space for host and pitched memory on the device.
    hipHostAlloc(&h_img, resX * resY * sizeof(int), hipHostMallocMapped);
    hipMallocPitch((int **)&d_img, &pitch, sizeof(int) * resX, resY);
    
    // Each thread processes 2*2 pixels.
    int thre_width = 2, thre_height = 2;

    // Based on the website "the CUDA programming guide" plus the CUDA version is V11.5, we have at most 1024 threads per block.
    dim3 blockSize(40, 25);
    dim3 numBlock(resX / (blockSize.x * thre_width), resY / (blockSize.y * thre_height));
    mandelKernel<<<numBlock, blockSize>>>(stepX, stepY, lowerX, lowerY, thre_width, thre_height, d_img, maxIterations, pitch);
    
    hipDeviceSynchronize();

    // Copy the result back to host.
    hipMemcpy2D(h_img, sizeof(int) * resX, d_img, pitch, sizeof(int) * resX, resY, hipMemcpyDeviceToHost);
    memcpy(img, h_img, resX * resY * sizeof(int));
    
    hipFree(d_img);
    hipHostFree(h_img);
    return;
}