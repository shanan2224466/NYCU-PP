#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float stepX, float stepY, float lowerX, float lowerY, int* d_img, int maxIterations, int pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;

    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < maxIterations; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    d_img[thisX + thisY * pitch / 4] = i;
    return;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *d_img;
    int *h_img;
    size_t pitch;

    // Allocate pinned memory space for host and pitched memory on the device.
    hipHostAlloc(&h_img, resX * resY * sizeof(int), hipHostMallocMapped);
    hipMallocPitch((int **)&d_img, &pitch, sizeof(int) * resX, resY);
    
    // Based on the website "the CUDA programming guide" plus the CUDA version is V11.5, we have at most 1024 threads per block.
    dim3 blockSize(40, 25);
    dim3 numBlock(resX / blockSize.x, resY / blockSize.y);
    mandelKernel<<<numBlock, blockSize>>>(stepX, stepY, lowerX, lowerY, d_img, maxIterations, pitch);
    
    hipDeviceSynchronize();

    // Copy the result back to host.
    hipMemcpy2D(h_img, sizeof(int) * resX, d_img, pitch, sizeof(int) * resX, resY, hipMemcpyDeviceToHost);
    memcpy(img, h_img, resX * resY * sizeof(int));
    
    hipFree(d_img);
    hipHostFree(h_img);
    return;
}
