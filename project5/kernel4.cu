#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float stepX, float stepY, float lowerX, float lowerY, int* d_img, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;

    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < maxIterations; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    d_img[thisX + thisY * gridDim.x * blockDim.x] = i;
}


// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *d_img;

    // Allocate memory space for host and device.
    hipMalloc((void **)&d_img, resX * resY * sizeof(int));

    // Based on the website "the CUDA programming guide" plus the CUDA version is V11.5, we have at most 1024 threads per block.
    dim3 blockSize(16, 12);
    dim3 numBlock(resX / blockSize.x, resY / blockSize.y);
    mandelKernel <<<numBlock, blockSize>>> (stepX, stepY, lowerX, lowerY, d_img, maxIterations);

    // Copy the result back to host.
    hipMemcpy(img, d_img, sizeof(int) * resX * resY, hipMemcpyDeviceToHost);

    hipFree(d_img);
}
